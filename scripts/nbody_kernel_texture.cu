
#include <hip/hip_runtime.h>
__constant__ double g;
__constant__ double time_step;
__constant__ double particle_count;

extern "C" {

__global__ void update_velocity( double * const __restrict__ mass, double *x , double *y , double *z , double *v_x , double *v_y , double *v_z ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < particle_count ) {
        double local_mass = mass[i];

        // Do force, acceleration, velocity, and movement effects for each particle
        for ( int j = 0 ; j < particle_count - 1 ; j++) {
            if( i != j ) {
                // Calculate total distance from two particles
                double dist = sqrt( pow( (double)(x[i] - x[j]) , (double)2 ) + pow( (double)(y[i] - y[j]) , (double)2 ) + pow( (double)(z[i] - z[j]) , (double)2 ) );

                // Calculate net force on two particles
                double net_force = ( ( local_mass * mass[j] ) / pow( dist , (double)2 ) ) * g;

                // Compute each force vector
                double f_x = net_force * ( ( x[j] - x[i] ) / dist );
                double f_y = net_force * ( ( y[j] - y[i] ) / dist );
                double f_z = net_force * ( ( z[j] - z[i] ) / dist );

                // Calculate new velocity with decay in consideration plus new acceleration relative to time_step
                v_x[i] = v_x[i] + ( f_x / local_mass ) * time_step;
                v_y[i] = v_y[i] + ( f_y / local_mass ) * time_step;
                v_z[i] = v_z[i] + ( f_z / local_mass ) * time_step;
            }
        }
    }
}

__global__ void update_position( double *x , double *y , double *z , double *v_x , double *v_y , double *v_z ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Update each coordinate location with current velocity in consideration over the span of a time time_step
    if ( i < particle_count ) {
        x[i] = x[i] + v_x[i] * time_step;
        y[i] = y[i] + v_y[i] * time_step;
        z[i] = z[i] + v_z[i] * time_step;
    }
}

}