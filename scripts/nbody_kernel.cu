
#include <hip/hip_runtime.h>
__constant__ unsigned int g;
__constant__ unsigned int time_step;
__constant__ unsigned int particle_count;

extern "C" {

__global__ void update_velocity( int *mass, float *x , float *y , float *z , float *v_x , float *v_y , float *v_z ) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < particle_count ) {
        int local_mass = mass[i];

        // Do force, acceleration, velocity, and movement effects for each particle
        for ( int j = 0 ; j < particle_count - 1 ; j++) {
            if( i != j ) {
                // Calculate total distance from two particles
                float dist = sqrt( pow( (float)(x[i] - x[j]) , (float)2 ) + pow( (float)(y[i] - y[j]) , (float)2 ) + pow( (float)(z[i] - z[j]) , (float)2 ) );

                // Calculate net force on two particles
                float net_force = ( ( local_mass * mass[j] ) / pow( dist , (float)2 ) ) * g;

                // Compute each force vector
                float f_x = net_force * ( ( x[j] - x[i] ) / dist );
                float f_y = net_force * ( ( y[j] - y[i] ) / dist );
                float f_z = net_force * ( ( z[j] - z[i] ) / dist );

                // Calculate new velocity with decay in consideration plus new acceleration relative to time_step
                v_x[i] = v_x[i] + ( f_x / local_mass ) * time_step;
                v_y[i] = v_y[i] + ( f_y / local_mass ) * time_step;
                v_z[i] = v_z[i] + ( f_z / local_mass ) * time_step;
            }
        }
    }
}

__global__ void update_position( float *x , float *y , float *z , float *v_x , float *v_y , float *v_z ) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Update each coordinate location with current velocity in consideration over the span of a time time_step
    if ( i < particle_count ) {
        x[i] = x[i] + v_x[i] * time_step;
        y[i] = y[i] + v_y[i] * time_step;
        z[i] = z[i] + v_z[i] * time_step;
    }
}

}